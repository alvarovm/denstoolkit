#include "hip/hip_runtime.h"
/*
 *  wavefunctionclass.cpp

 
 
 ------------------------
 
 Juan Manuel Solano Altamirano
 Adscription at the moment this project is initiated:
 Department of Chemistry, University of Guelph,
 Guelph, Ontario, Canada.
 e-mail: jmsolanoalt@gmail.com
 
 ------------------------
 
 This code is free code; you can redistribute it and/or
 modify it under the terms of the GNU General Public License
 as published by the Free Software Foundation; either version 2
 of the License, or (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software 
 Foundation, Inc., 59 Temple Place - Suite 330, 
 Boston, MA  02111-1307, USA.
 
 WWW:  http://www.gnu.org/copyleft/gpl.html
 
 ----------------------
 
 */

#ifndef _SOLWAVEFUNCTIONCLASS_CPP_
#define _SOLWAVEFUNCTIONCLASS_CPP_

#include "cudawavefunctionclass.cuh"
#include "solmemhand.cpp"
#include "iofuncts-wfn.cpp"
#include "iofuncts-wfx.cpp"
#include "eig2-4.cpp"

#ifndef DEBUG
#define DEBUG 0
#endif

#ifndef EPSFORELFVALUE
#define EPSFORELFVALUE (2.871e-05)
#endif

#ifndef EPSFORLOLVALUE
#define EPSFORLOLVALUE (2.871e-05)
#endif

#ifndef BASETHREADSPERBLOCK
#define BASETHREADSPERBLOCK 16
#endif

//*************************************************************************************************
//*************************************************************************************************
//*************************************************************************************************
//*************************************************************************************************
gaussWaveFunc::gaussWaveFunc()
//*************************************************************************************************
{
   title=NULL;
   orbDesc=string("");
   nTit=0;
   nNuc=0;
   nMOr=0;
   nPri=0;
   atLbl=NULL;
   primType=NULL;
   primCent=NULL;
   myPN=NULL;
   R=NULL;
   atCharge=NULL;
   primExp=NULL;
   MOCoeff=NULL;
   occN=NULL;
   MOEner=NULL;
   cab=NULL;
   chi=NULL;
   gx=gy=gz=NULL;
   hxx=hyy=hzz=NULL;
   hxy=hxz=hyz=NULL;
   totener=0.00e0;
   virial=0.0e0;
   imldd=false;
   h_R=d_R=d_e=d_c=NULL;
   h_a=d_a=NULL;
   d_chi=d_rho=d_aux=NULL;
}
//*************************************************************************************************
int gaussWaveFunc::prTy[]={
   0, 0, 0,   1, 0, 0,   0, 1, 0,   0, 0, 1,   2, 0, 0, 
   0, 2, 0,   0, 0, 2,   1, 1, 0,   1, 0, 1,   0, 1, 1, 
   3, 0, 0,   0, 3, 0,   0, 0, 3,   1, 2, 0,   2, 1, 0, 
   2, 0, 1,   1, 0, 2,   0, 1, 2,   0, 2, 1,   1, 1, 1
};
//*************************************************************************************************
gaussWaveFunc::~gaussWaveFunc()
//*************************************************************************************************
{
   dealloc1DStringArray(title);
   dealloc1DRealArray(R);
   dealloc1DStringArray(atLbl);
   dealloc1DRealArray(atCharge);
   dealloc1DIntArray(primCent);
   dealloc1DIntArray(primType);
   dealloc1DRealArray(primExp);
   dealloc1DRealArray(chi);
   dealloc1DRealArray(cab);
   dealloc1DRealArray(MOCoeff);
   dealloc1DRealArray(occN);
   dealloc1DRealArray(MOEner);
   dealloc1DRealArray(gx);
   dealloc1DRealArray(gy);
   dealloc1DRealArray(gz);
   dealloc1DRealArray(hxx);
   dealloc1DRealArray(hyy);
   dealloc1DRealArray(hzz);
   dealloc1DRealArray(hxy);
   dealloc1DRealArray(hxz);
   dealloc1DRealArray(hyz);
   cleanWaveFunctionInGPU(&h_R,&d_R,&h_a,&d_a,&d_e,&d_c,&d_chi,&d_rho,&d_aux);
   imldd=false;
}
//*************************************************************************************************
bool gaussWaveFunc::readFromFileWFN(string inname)
//*************************************************************************************************
{
   ifstream tif;
   tif.open(inname.c_str(),ios::in);
   if (!(tif.good())) {
      cout << "Error: File " << inname << "could not be opened...\n";
#if DEBUG
      cout << __FILE__ << ", line: " << __LINE__ << endl;
#endif
      return false;
   }
   tif.seekg(tif.beg);
   nTit=1;
   processFirstDataStringinWFNFile(tif,title,orbDesc,nMOr,nPri,nNuc);
   processCentersWFN(tif,nNuc,atLbl,R,atCharge);
   processPrimitivesWFN(tif,nPri,primCent,primType,primExp);
   processMolecularOrbitalPropsAndCoefs(tif,nMOr,nPri,occN,MOEner,MOCoeff);
   string liend;
   getline(tif,liend);
   //cout << "nPri%5: " << (nPri%5) << " len: " << liend.length() << endl;
   if (((nPri%5)==0)&&(liend.length()==0)) {
      getline(tif,liend);
      cout << liend << endl;
   }
   if (liend.substr(0,8)!="END DATA") {
      cout << "Error, expecting \"END DATA\" in file " << inname << endl;
      cout << "Line: " << liend << endl;
      return false;
   }
   getEnergyAndVirial(tif,totener,virial);
   allocAuxArrays();
   countPrimsPerCenter();
   calcCab();
   tif.close();
   imldd=testSupport();
   return true;
}
//*************************************************************************************************
bool gaussWaveFunc::readFromFileWFX(string inname)
//*************************************************************************************************
{
   ifstream tif;
   tif.open(inname.c_str(),ios::in);
   if (!(tif.good())) {
      cout << "Error: File " << inname << "could not be opened...\n";
#if DEBUG
      cout << __FILE__ << ", line: " << __LINE__ << endl;
#endif
      return false;
   }
   getTitleFromFileWFX(tif,nTit,title);
   getKeyWordsFromFileWFX(tif,orbDesc);
   if (orbDesc.substr(0,3)!="GTO") {
      cout << "Error: not supported wave function. Keyword: " << orbDesc << endl;
   }
   getNofNucleiFromFileWFX(tif,nNuc);
   getNofMolOrbFromFileWFX(tif,nMOr);
   getNofPrimFromFileWFX(tif,nPri);
   alloc1DStringArray("atLbl",nNuc,atLbl);
   alloc1DIntArray("primType",nPri,primType);
   alloc1DIntArray("primCent",nPri,primCent);
   alloc1DRealArray("R",(3*nNuc),R);
   alloc1DRealArray("atCharge",nNuc,atCharge);
   alloc1DRealArray("primExp",nPri,primExp);
   alloc1DRealArray("MOCoeff",(nMOr*nPri),MOCoeff);
   alloc1DRealArray("occN",nMOr,occN);
   alloc1DRealArray("MOEner",nMOr,MOEner);
   allocAuxArrays();
   getAtLabelsFromFileWFX(tif,nNuc,atLbl);
   getNucCartCoordsFromFileWFX(tif,nNuc,R);
   getAtChargesFromFileWFX(tif,nNuc,atCharge);
   getPrimCentersFromFileWFX(tif,nPri,primCent);
   getPrimTypesFromFileWFX(tif,nPri,primType);
   getPrimExponentsFromFileWFX(tif,nPri,primExp);
   getMolecOrbOccNumsFromFileWFX(tif,nMOr,occN);
   getMolecOrbEnergiesFromFileWFX(tif,nMOr,MOEner);
   getMolecOrbCoefficientsFromFileWFX(tif,nMOr,nPri,MOCoeff);
   getTotEnerAndVirialFromFileWFX(tif,totener,virial);
   countPrimsPerCenter();
   calcCab();
   tif.close();
   imldd=testSupport();
   return true;
}
//*************************************************************************************************
bool gaussWaveFunc::readFromFile(string inname)
{
   string extension;
   extension=inname.substr(inname.length()-3,3);
   if ((extension=="wfn")||(extension=="WFN")) {
      return readFromFileWFN(inname);
   } else if ((extension=="wfx")||(extension=="WFX")) {
      return readFromFileWFX(inname);
   } else {
      cout << "Error: unknown extension ("  << inname << ")!\nNothig to do, returning false...\n";
      cout << __FILE__ << ", line: " << __LINE__ << endl;
      return false;
   }
}
//*************************************************************************************************
bool gaussWaveFunc::testSupport()
{
   for (int i=0; i<nMOr; i++) {
      if (primType[i]>=MAXPRIMTYPEDEFINED) {
         cout << "Only " << MAXPRIMTYPEDEFINED << " types have been implemented in this version\n";
#if DEBUG
         cout << __FILE__ << "line " << __LINE__ << endl;
#endif
         return false;
      }
   }
   return true;
}
//*************************************************************************************************
void gaussWaveFunc::calcCab(void)
{
   int idx,indc;
   idx=0;
   if (nPri>MAXNUMBEROFPRIMITIVESFORMEMALLOC) {
      real memest=real(nPri*(nPri+12)*8)/real(1024*1024);
      char goon='n';
      cout << "The number of primitives is " << nPri <<". This will use approximatedly" << endl;
      cout << memest << "MB of RAM memory. Continue anyway (y/n)?" << endl;
      cin >> goon;
      if ((goon=='n')||(goon=='N')) {
         cout << "Perhaps you may want to recompile this program increasing the maximum number " << endl
              << "  of primitives. " << endl;
         exit(1);
      }
   }
   alloc1DRealArray(string("cab"),(nPri*nPri),cab);
   for (int i=0; i<nPri; i++) {
      for (int j=0; j<nPri; j++) {
         cab[idx]=0.0000000e0;
         for (int oi=0; oi<nMOr; oi++) {
            indc=oi*nPri;
            cab[idx]+=(occN[oi]*MOCoeff[indc+i]*MOCoeff[indc+j]);
         }
         idx++;
      }
   }
   return;
}
//*************************************************************************************************
void gaussWaveFunc::countPrimsPerCenter(void)
{
   alloc1DIntArray(string("myPN"),nNuc,myPN);
   for (int i=0; i<nPri; i++) {myPN[primCent[i]]++;}
   return;
}
//*************************************************************************************************
bool gaussWaveFunc::allocAuxArrays(void)
{
   bool allgood;
   allgood=alloc1DRealArray("chi",nPri,chi);
   if (!allgood) {
      cout << "Something wrong in allocating chi..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("gx"),nPri,gx);
   if (!allgood) {
      cout << "Something wrong in allocating gx..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("gy"),nPri,gy);
   if (!allgood) {
      cout << "Something wrong in allocating gy..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("gz"),nPri,gz);
   if (!allgood) {
      cout << "Something wrong in allocating gz..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("hxx"),nPri,hxx);
   if (!allgood) {
      cout << "Something wrong in allocating hxx..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("hyy"),nPri,hyy);
   if (!allgood) {
      cout << "Something wrong in allocating hyy..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("hzz"),nPri,hzz);
   if (!allgood) {
      cout << "Something wrong in allocating hzz..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("hxy"),nPri,hxy);
   if (!allgood) {
      cout << "Something wrong in allocating hxy..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("hxz"),nPri,hxz);
   if (!allgood) {
      cout << "Something wrong in allocating hxz..." << endl;
      return allgood;
   }
   allgood=alloc1DRealArray(string("hyz"),nPri,hyz);
   if (!allgood) {
      cout << "Something wrong in allocating hyz..." << endl;
      return allgood;
   }
   return allgood;
}
//*************************************************************************************************
extern "C" void setupWaveFunctionInGPU(int npr, \
                                       real** host_R,real** dev_R, \
                                       int ** host_a,int ** dev_a, \
                                                     real** dev_e, \
                                                     real** dev_c, \
                                       real** dev_chi,real** dev_rho,real** dev_aux)
{
   //printf("Entering allocation function...\n");
   unsigned int mem_size=sizeof(real)*3*npr;
   *host_R=(real*)malloc(mem_size);
   checkCudaErrors(hipMalloc((void **) &(*dev_R), mem_size));
   mem_size=sizeof(int)*3*npr;
   *host_a=(int*)malloc(mem_size);
   checkCudaErrors(hipMalloc((void **) &(*dev_a), mem_size));
   mem_size=sizeof(real)*npr;
   checkCudaErrors(hipMalloc((void **) &(*dev_e), mem_size));
   mem_size=sizeof(real)*npr*npr;
   checkCudaErrors(hipMalloc((void **) &(*dev_c), mem_size));
   mem_size=sizeof(real)*npr;
   checkCudaErrors(hipMalloc((void **) &(*dev_chi), mem_size));
   checkCudaErrors(hipMalloc((void **) &(*dev_aux), mem_size));
   checkCudaErrors(hipMalloc((void **) &(*dev_rho), sizeof(real)));
   //printf("Done\n");
   return;
}
//**************************************************************************************************
extern "C" void cleanWaveFunctionInGPU(real **host_R,real** dev_R,\
                                       int **host_a,int **dev_a, \
                                                    real**dev_e, \
                                                    real**dev_c, \
                                       real**dev_chi,real**dev_rho,real**dev_aux)
{
   if (*host_R!=NULL) {
      free(*host_R);
      *host_R=NULL;
   }
   checkCudaErrors(hipFree(*dev_R));
   if (*host_a!=NULL) {
      free(*host_a);
      *host_a=NULL;
   }
   checkCudaErrors(hipFree(*dev_a));
   checkCudaErrors(hipFree(*dev_e));
   checkCudaErrors(hipFree(*dev_c));
   checkCudaErrors(hipFree(*dev_chi));
   checkCudaErrors(hipFree(*dev_aux));
   checkCudaErrors(hipFree(*dev_rho));
   return;
}
//**************************************************************************************************
bool gaussWaveFunc::setupGPU(void)
{
   //cout << "Checkpoint..." << endl;
   //cout << "h_R: " << h_R << ", d_R: " << d_R << endl;
   //cout << "h_a: " << h_a << ", d_a: " << d_a << endl;
   setupWaveFunctionInGPU(nPri,&h_R,&d_R,&h_a,&d_a,&d_e,&d_c,&d_chi,&d_rho,&d_aux);
   //cout << "h_R: " << h_R << ", d_R: " << d_R << endl;
   //cout << "h_a: " << h_a << ", d_a: " << d_a << endl;
   size_t idc,idr,idp;
   for (size_t i=0; i<nPri; i++) {
      idc=primCent[i]*3;
      idr=i*3;
      for (size_t j=0; j<3; j++) {
         h_R[idr+j]=R[idc+j];
      }
      idc=primType[i]*3;
      idp=i*3;
      for (size_t j=0; j<3; j++) {
         h_a[idp+j]=prTy[idc+j];
      }
   }
   copyDataToGPU(nPri,&h_R,&d_R,&h_a,&d_a,&primExp,&d_e,&cab,&d_c);
   //dosomethinginGPU(nPri,&d_R,&d_a,&d_e,&d_c);
   //copyDataFromGPU(nPri,&d_R,&h_R,&d_a,&h_a,&d_e,&primExp,&d_c,&cab);
   return true;
}
//**************************************************************************************************
real gaussWaveFunc::evalDensity(real x,real y,real z)
{
   return evalDensityInGPU(x,y,z,nPri,&d_R,&d_a,&d_e,&d_c,&chi,&d_chi,&d_rho);
}
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
extern "C" void copyDataToGPU(int npr,real **host_R,real **dev_R,int **host_a,int **dev_a, \
                                      real **host_e,real **dev_e,real**host_c,real**dev_c)
{
   unsigned int mem_size=sizeof(real)*3*npr;
   checkCudaErrors(hipMemcpy(*dev_R, *host_R, mem_size, hipMemcpyHostToDevice));
   mem_size=sizeof(int)*3*npr;
   checkCudaErrors(hipMemcpy(*dev_a, *host_a, mem_size, hipMemcpyHostToDevice));
   mem_size=sizeof(real)*npr;
   checkCudaErrors(hipMemcpy(*dev_e, *host_e, mem_size, hipMemcpyHostToDevice));
   mem_size=sizeof(real)*npr*npr;
   checkCudaErrors(hipMemcpy(*dev_c, *host_c, mem_size, hipMemcpyHostToDevice));
   return;
}
//**************************************************************************************************
extern "C" void copyDataFromGPU(int npr,real **dev_R,real **host_R,int **dev_a,int **host_a, \
                                        real **dev_e,real **host_e,real**dev_c,real**host_c)
{
   unsigned int mem_size=sizeof(real)*3*npr;
   checkCudaErrors(hipMemcpy(*host_R, *dev_R, mem_size,hipMemcpyDeviceToHost));
   mem_size=sizeof(int)*3*npr;
   checkCudaErrors(hipMemcpy(*host_a, *dev_a, mem_size,hipMemcpyDeviceToHost));
   mem_size=sizeof(real)*npr;
   checkCudaErrors(hipMemcpy(*host_e, *dev_e, mem_size,hipMemcpyDeviceToHost));
   mem_size=sizeof(real)*npr*npr;
   checkCudaErrors(hipMemcpy(*host_c, *dev_c, mem_size,hipMemcpyDeviceToHost));
   return;
}
//**************************************************************************************************
__global__ void kernelR(int rdim,real *dev_R)
{
   unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;;
   if (tid<rdim) {
      dev_R[tid]=1.0e0+0.1*real(tid);
   }
   return;
}
//**************************************************************************************************
__global__ void kernela(int rdim,int *dev_a)
{
   unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
   if (tid<rdim) {
      //dev_a[tid]=tid;
      dev_a[tid]+=10;
   }
   return;
}
//**************************************************************************************************
__global__ void kernele(int rdim,real *dev_e)
{
   unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
   if (tid<rdim) {
      dev_e[tid]*=(-1);
   }
   return;
}
//**************************************************************************************************
__global__ void kernelc(int rdim,real *dev_c)
{
   unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
   if (tid<rdim) {
      dev_c[tid]*=(-1);
   }
   return;
}
//**************************************************************************************************
__global__ void krnlCalcChi(int rdim,real *dev_R,int *dev_a,real *dev_e,real *dev_chi, \
                            real x,real y,real z)
{
   unsigned int btid = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int ttid = 3*btid;
   real r2,xmr,ymr,zmr,ang;
   if (btid<rdim) {
      xmr=x-dev_R[ttid];
      ymr=y-dev_R[ttid+1];
      zmr=z-dev_R[ttid+2];
      r2=(xmr*xmr)+(ymr*ymr)+(zmr*zmr);
      ang=1.0e0;
      for (int i=0; i<dev_a[ttid]; i++) {
         ang*=xmr;
      }
      for (int i=0; i<dev_a[ttid+1]; i++) {
         ang*=ymr;
      }
      for (int i=0; i<dev_a[ttid+2]; i++) {
         ang*=zmr;
      }
      dev_chi[btid]=ang*__expf(-r2*dev_e[btid]);
   }
   return;
}
//**************************************************************************************************
__global__ void krnlCalcRho(int rdim,int offset,real *dev_c,real *dev_chi,real *dev_rho)
{
   __shared__ real tmp[BASETHREADSPERBLOCK];
   unsigned int ptid = blockDim.x * blockIdx.x + threadIdx.x;
   
   if (ptid<rdim) {
      tmp[threadIdx.x]=dev_chi[ptid]*dev_c[offset + ptid];
   } else {
      tmp[threadIdx.x]=0.0e0;
   }
   __syncthreads();
   //if ( ptid == 0 ) {*dev_rho=0.0e0;}
   if (threadIdx.x == 0) {
      real sum=0.0e0;
      for (unsigned int j=0; j<BASETHREADSPERBLOCK; j++) {
         sum+=tmp[j];
      }
      //*dev_rho+=sum;
      atomicAdd(dev_rho,sum);
   }
}
//**************************************************************************************************
__device__ double atomicAdd(double* address, double val)
{
   unsigned long long int* address_as_ull =
   (unsigned long long int*)address;
   unsigned long long int old = *address_as_ull, assumed;
   do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed,
                      __double_as_longlong(val +
                                           __longlong_as_double(assumed)));
   } while (assumed != old);
   return __longlong_as_double(old);
}
//**************************************************************************************************
//**************************************************************************************************
extern "C" void dosomethinginGPU(int npr,real **dev_R,int **dev_a,real **dev_e,real **dev_c)
{
   int numElements=3*npr;
   int threadsPerBlock = BASETHREADSPERBLOCK;
   int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
   //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
   //printf("numElements= %d\n",numElements);
   kernelR<<<blocksPerGrid, threadsPerBlock>>>(numElements,*dev_R);
   kernela<<<blocksPerGrid, threadsPerBlock>>>(numElements,*dev_a);
   numElements=npr;
   blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
   kernele<<<blocksPerGrid, threadsPerBlock>>>(numElements,*dev_e);
   numElements=npr*npr;
   blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
   kernelc<<<blocksPerGrid, threadsPerBlock>>>(numElements,*dev_c);
   return;
}
//**************************************************************************************************
extern "C" real evalDensityInGPU(real x,real y,real z,
                                 int npr,real **dev_R,int **dev_a,real **dev_e,real **dev_c, \
                                 real **host_chi,real **dev_chi,real **dev_rho)
{
   int numElements=npr;
   int threadsPerBlock = BASETHREADSPERBLOCK;
   int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
   krnlCalcChi<<<blocksPerGrid, threadsPerBlock>>>(numElements,*dev_R,*dev_a,*dev_e,*dev_chi, \
                                                   x,y,z);
   unsigned int mem_size=sizeof(real)*npr;
   checkCudaErrors(hipMemcpy(*host_chi, *dev_chi, mem_size, hipMemcpyDeviceToHost));
   real rho=0.0e0,trho;
   int offst;
   for (int i=0; i<npr; i++) {
      offst=i*npr;
      trho=0.0e0;
      checkCudaErrors(hipMemcpy(*dev_rho, &trho, sizeof(real), hipMemcpyHostToDevice));
      krnlCalcRho<<<blocksPerGrid, threadsPerBlock>>>(numElements,offst,*dev_c,*dev_chi,*dev_rho);
      checkCudaErrors(hipMemcpy(&trho, *dev_rho, sizeof(real), hipMemcpyDeviceToHost));
      rho+=trho*((*host_chi)[i]);
   }
   return rho;
   //unsigned int mem_size=sizeof(real)*npr;
   //checkCudaErrors(hipMemcpy(*host_chi, *dev_chi, mem_size,hipMemcpyDeviceToHost));
}
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
//**************************************************************************************************
#endif//_SOLWAVEFUNCTIONCLASS_CPP_

